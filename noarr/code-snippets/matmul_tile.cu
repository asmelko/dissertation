#include "hip/hip_runtime.h"
template <char I, char J, class struct_lhs_t, class struct_rhs_t, class struct_out_t>
__global__ float matmul_tile(const float* lhs_in, const float* rhs_in, float* out, const struct_lhs_t lhs_s, const struct_rhs_t rhs_s, struct_out_t out_s) {
    constexpr size_t tile_w = 16;
    constexpr auto tile_s = #\noarr{}#::array<I, tile_w, #\noarr{}#::array<J, tile_w, #\noarr{}#::scalar<float>>>();
    __shared__ float l_tile[tile_w * tile_w];
    __shared__ float r_tile[tile_w * tile_w];
    const uint32_t x = blockIdx.x * tile_size + threadIdx.x;
    const uint32_t y = blockIdx.y * tile_size + threadIdx.y;

    float acc = 0.f;
    for (uint32_t i = 0; i < lhs_s.get_length<J>(); i += tile_w) {
        tile_s.get_at<I, J>(l_tile, threadIdx.y, threadIdx.x) =
            lhs_s.get_at<I, J>(lhs_data, y, threadIdx.x + i);
        tile_s.get_at<I, J>(r_tile, threadIdx.y, threadIdx.x) =
            rhs_s.get_at<I, J>(rhs_data, threadIdx.y + i, x);
        __syncthreads();

        for (uint32_t j = 0; j < tile_w; j++)
            acc += tile_s.get_at<I, J>(l_tile, threadIdx.y, j)
                   * tile_s.get_at<J, I>(r_tile, threadIdx.x, j);
        __syncthreads();
    }
    out_s.get_at<I, J>(output_data, y, x) = acc;
}